#include "hip/hip_runtime.h"
static const char RCSsetup_c[] = "$Id: setup.c,v 8.1 2007/06/23 22:33:35 wedgingt Exp $";

#include "setup.h"

static const char RCSsetup_h[] = DRCSsetup_h;

#ifndef vms
volatile char shouldTerminate = 0; /* Flag: have we gotten a SIGTERM ? */
#endif

/* only used to make the process's priority as low as possible */
//#if !defined(macintosh) && !defined(_MSC_VER)
//extern int nice (int priority_increment);
//#endif

#ifdef __sgi__
#include <limits.h>
#include <sys/types.h>
#include <sys/prctl.h>
#include <sys/schedctl.h>
#endif

/* same as the BSD index() and the SYSV strchr() */
const char *presence(const char *string, int ch)
{
  if (string == NULL)
    return(NULL);
  while (*string != '\0')
    if (*string == ch)
      return(string);
    else
      ++string;
  return(NULL);
}

void setup (void)
 {
  setlinebuf(stdout);
//  setlinebuf(stderr);
#ifdef __sgi__
  (void)schedctl(NDPRI, 0, NDPLOMIN);
#endif
#if !defined(macintosh) && !defined(_MSC_VER)
//  (void)nice(40);
#endif
#ifdef SIGTERM
  (void)signal(SIGTERM, term_handler);
#endif
#ifdef SIGINT
  (void)signal(SIGINT, term_handler);
#endif
#ifdef SIGHUP
  (void)signal(SIGHUP, term_handler);
#endif
#ifdef SIGPIPE
  (void)signal(SIGPIPE, SIG_IGN);
#endif
#ifdef ANCIENT_linux
  /*!!Should not be needed any more, as implied by the new macro protecting it.  Old comments: */
  /*!!really ought to examine the signal and let it happen if it's an overflow, NaN, etc.,*/
  /*!! but it's almost always an underflow or round off warning rather than a problem like overflow or NaN */
  (void)signal(SIGFPE, SIG_IGN);
#endif
  return;
 }

#if !defined(vms)
/*ARGSUSED*/ /* they aren't but that's because only some OS's want an argument here */
# if defined(linux) || defined(__ultrix) || defined(_AIX) || defined(__hpux) || defined(macintosh) || defined(__APPLE__) || defined(_MSC_VER)
handler term_handler(int)
# else
handler term_handler()
# endif
 {
# ifdef SIGTERM
  (void)signal(SIGTERM, SIG_IGN);
# endif
# ifdef SIGINT
  (void)signal(SIGINT, SIG_IGN);
# endif
# ifdef SIGHUP
  (void)signal(SIGHUP, SIG_IGN);
# endif
  shouldTerminate = 1;
  return_handler;
}

# ifndef pc7300
void clientexit(const char *msg)
{
  (void)fprintf(stderr, "%s: ", program_name);
  (void)fflush(stderr);
  perror(msg);
  exit(errno);
}

#  ifdef NO_HERROR
#   ifndef NO_ADDRESS
#    define NO_ADDRESS 3
#   endif
void herror(const char *msg)
{
  static const char *(herrs[]) = {
    "unknown error",
    "no such host",
    "server failure or host not known",
    "no address for host"
  };

  (void)fprintf(stderr, "%s: %s (code %d)\n", msg, h_errno <= NO_ADDRESS ? herrs[h_errno] : herrs[0],
        h_errno);
}
#  endif
# endif
#endif
